#include "hip/hip_runtime.h"

#include <hipsolver.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <>

#include "solver.h"


__global__ void cuSpPrecondition(double* diagonal_inv, double* data, long long int n) {
    long long int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        double d = data[idx];
        if (diagonal_inv[idx] != 0)
            data[idx] = d / diagonal_inv[idx];
        else
            data[idx] = d * 0.1;
    }
}

__global__ void cuProject(double* data, long long int n) {
    long long int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        double d = data[idx];
        if (d < 0)
            data[idx] = 0;
    }
}

void _amg_gmres(double* csrval, int* csrcol, int* csrrow, int n, int nnz, double* yb, double* dx) {
    AMGX_SAFE_CALL(AMGX_initialize());
    AMGX_SAFE_CALL(AMGX_initialize_plugins());
    AMGX_config_handle cfg;
    AMGX_SAFE_CALL(AMGX_config_create_from_file(&cfg, "E:/code/NanoporePDE/configs/FGMRES_CLASSICAL_AGGRESSIVE_PMIS.json"));
    AMGX_resources_handle rsrc;
    AMGX_SAFE_CALL(AMGX_resources_create_simple(&rsrc, cfg));
    
    AMGX_matrix_handle A;
    AMGX_vector_handle x;
    AMGX_vector_handle b;
    AMGX_solver_handle solver;

    AMGX_SAFE_CALL(AMGX_matrix_create(&A, rsrc, AMGX_mode_dDDI));
    AMGX_SAFE_CALL(AMGX_vector_create(&x, rsrc, AMGX_mode_dDDI));
    AMGX_SAFE_CALL(AMGX_vector_create(&b, rsrc, AMGX_mode_dDDI));
    AMGX_SAFE_CALL(AMGX_solver_create(&solver, rsrc, AMGX_mode_dDDI, cfg));

    AMGX_SAFE_CALL(AMGX_matrix_upload_all(A, n, nnz, 1, 1, csrrow, csrcol, csrval, nullptr));
    AMGX_SAFE_CALL(AMGX_vector_upload(b, n, 1, yb));
    AMGX_SAFE_CALL(AMGX_vector_upload(x, n, 1, dx));
    AMGX_SAFE_CALL(AMGX_solver_setup(solver, A));
    AMGX_SAFE_CALL(AMGX_solver_solve(solver, b, x));
    AMGX_SAFE_CALL(AMGX_vector_download(x, dx));
}

void _tfqmr(double* csrval, long long int* csrcol, long long int* csrrow, void** d_buffer, double* diag, long long int n, long long int nnz, bool* buffer,
    double* y, double* x, double* yk1, double* yk2, double* r0, double* uk1, double* uk2, double* wk, double* vk, double* rk, double* dk) {
    //sove J*dv = -F;
    dim3 griddim((n + 255) / 256);
    dim3 blockdim(256);
    double epsi = 0.0;
    double theta = 0.0;
    double rho;
    double tau;
    double normal_b;
    double rho_new;
    double const_none = -1.0;
    double const_one = 1.0;
    double const_zero = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipsparseHandle_t sphandle;
    hipsparseSpMatDescr_t matJ;
    hipsparseDnVecDescr_t vecX, vecY;
    hipsparseCreate(&sphandle);
    hipsparseCreateDnVec(&vecX, n, x, HIP_R_64F); // x
    hipsparseCreateDnVec(&vecY, n, r0, HIP_R_64F); // y 
    hipsparseCreateCsr(&matJ, n, n, nnz, csrrow, csrcol, csrval, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F); // J
    size_t bufferSize = 0;
    if (*buffer) {
        hipsparseSpMV_bufferSize(sphandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &const_none, matJ, vecX, &const_one, vecY, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize);
        hipMalloc(d_buffer, bufferSize);
        *buffer = false;
    }
    //hipMemset(x, 0.5, n * sizeof(double));
    hipblasDcopy(handle, n, y, 1, r0, 1);
    hipDeviceSynchronize();
    cuSpPrecondition << <griddim, blockdim >> > (diag, y, n);
    hipDeviceSynchronize();
    hipblasDnrm2(handle, n, y, 1, &normal_b);
    /* ==================start solve the linear equations ================*/
    // calculte r0 = yb - Ax 
    hipsparseSpMV(sphandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &const_none, matJ, vecX, &const_one, vecY, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, *d_buffer);
    hipDeviceSynchronize();
    cuSpPrecondition << <griddim, blockdim >> > (diag, r0, n);
    hipDeviceSynchronize();
    hipblasDcopy(handle, n, r0, 1, yk1, 1);
    hipblasDcopy(handle, n, r0, 1, wk, 1);
    hipblasDcopy(handle, n, r0, 1, rk, 1);
    hipMemset(dk, 0, n * sizeof(double));
    hipsparseDnVecSetValues(vecX, yk1);
    hipsparseDnVecSetValues(vecY, uk1);
    hipsparseSpMV(sphandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &const_one, matJ, vecX, &const_zero, vecY, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, *d_buffer);
    hipDeviceSynchronize();
    cuSpPrecondition << <griddim, blockdim >> > (diag, uk1, n);
    hipDeviceSynchronize();
    hipblasDcopy(handle, n, uk1, 1, vk, 1);
    hipblasDdot(handle, n, r0, 1, r0, 1, &rho);
    hipblasDnrm2(handle, n, rk, 1, &tau);
    hipDeviceSynchronize();
    // start;
    int k = 0;
    bool terminate = false;
    double residual;

    while (k < 20000 && !terminate) {
        k++;
        // 1. sigma = <r0,v>; alpha = rho / simga; y2 = y1 - alpha*v; u2 = Ay2;
        double sigma;
        hipblasDdot(handle, n, r0, 1, vk, 1, &sigma);
        hipDeviceSynchronize();
        double alpha = rho / sigma;
        double temporary = -alpha;
        hipblasDcopy(handle, n, yk1, 1, yk2, 1);
        hipblasDaxpy(handle, n, &temporary, vk, 1, yk2, 1);
        hipDeviceSynchronize();
        hipsparseDnVecSetValues(vecX, yk2);
        hipsparseDnVecSetValues(vecY, uk2);
        hipsparseSpMV(sphandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &const_one, matJ, vecX, &const_zero, vecY, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, *d_buffer);
        hipDeviceSynchronize();
        cuSpPrecondition << <griddim, blockdim >> > (diag, uk2, n);
        hipDeviceSynchronize();
        //2. odd or even
        for (int j = 1; j <= 2; j++) {
            int m = 2 * k - 2 + j;
            double coeff;
            if (j == 1) {
                hipblasDaxpy(handle, n, &temporary, uk1, 1, wk, 1);
                coeff = theta * theta * epsi / alpha;
                hipblasDscal(handle, n, &coeff, dk, 1);
                hipblasDaxpy(handle, n, &const_one, yk1, 1, dk, 1);
            }
            else {
                hipblasDaxpy(handle, n, &temporary, uk2, 1, wk, 1);
                coeff = theta * theta * epsi / alpha;
                hipblasDscal(handle, n, &coeff, dk, 1);
                hipblasDaxpy(handle, n, &const_one, yk2, 1, dk, 1);
            }
            hipblasDnrm2(handle, n, wk, 1, &theta);
            hipDeviceSynchronize();
            theta = theta / tau;
            double c = 1 / sqrt(1 + theta * theta);
            tau = tau * theta * c;
            epsi = c * c * alpha;
            hipblasDaxpy(handle, n, &epsi, dk, 1, x, 1);
            hipDeviceSynchronize();
            residual = tau * sqrt(double(m + 1)) / normal_b;
            if (k % 1 == 0 && j == 1)
                std::cout << "step: " << k << " tfres: " << residual << " rb: " << normal_b << std::endl;
            if (residual <= 1e-4 || rho == 0) {
                terminate = true;
                if (rho == 0)
                    std::cout << "rho 0 stop" << std::endl;
                else {
                    std::cout << "step: " << k << " tfres: " << residual << " rb: " << normal_b << std::endl;
                }
                break;
            }
        }
        //3. rho = <r0, w>; beta = rho / rho_old
        double rho_new;
        hipblasDdot(handle, n, r0, 1, wk, 1, &rho_new);
        hipDeviceSynchronize();
        double beta = rho_new / rho;
        rho = rho_new;
        //4. y1 = w + beta*y2, u1 = Ay1;
        hipblasDcopy(handle, n, yk2, 1, yk1, 1);
        hipblasDscal(handle, n, &beta, yk1, 1);
        hipblasDaxpy(handle, n, &const_one, wk, 1, yk1, 1);
        hipDeviceSynchronize();
        hipsparseDnVecSetValues(vecX, yk1);
        hipsparseDnVecSetValues(vecY, uk1);
        hipsparseSpMV(sphandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &const_one, matJ, vecX, &const_zero, vecY, HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, *d_buffer);
        hipDeviceSynchronize();
        cuSpPrecondition << <griddim, blockdim >> > (diag, uk1, n);
        hipDeviceSynchronize();
        //5. v = u1 + beta(u2 + beta*v);
        hipblasDscal(handle, n, &beta, vk, 1);
        hipblasDaxpy(handle, n, &const_one, uk2, 1, vk, 1);
        hipblasDscal(handle, n, &beta, vk, 1);
        hipblasDaxpy(handle, n, &const_one, uk1, 1, vk, 1);
        hipDeviceSynchronize();
    }
    //std::cout << k << " " << residual << std::endl;
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroySpMat(matJ);
    hipblasDestroy(handle);
    hipsparseDestroy(sphandle);
}


void PNPNSSolver::solve(PNPNS::CSRMatrix& A, double* x, double* yc, long long int nn, long long int nnz) {
    /*
    allocate the memory on gpu for tfqmr
    */
    double* csrval_gpu;
    long long int* csrcol_gpu;
    long long int* csrrow_gpu;
    double* x_gpu;
    double* yc_gpu;
    double* yk1_gpu;
    double* yk2_gpu;
    double* r0_gpu;
    double* uk1_gpu;
    double* uk2_gpu;
    double* wk_gpu;
    double* vk_gpu;
    double* rk_gpu;
    double* dk_gpu;
    double* diaginv_gpu;

    hipMalloc((void**)&csrval_gpu, nnz * sizeof(double));
    hipMalloc((void**)&csrcol_gpu, nnz * sizeof(long long  int));
    hipMalloc((void**)&csrrow_gpu, (nn + 1) * sizeof(long long int));
    hipMalloc((void**)&x_gpu, nn * sizeof(double));
    hipMalloc((void**)&yc_gpu, nn * sizeof(double));
    hipMalloc((void**)&yk1_gpu, nn * sizeof(double));
    hipMalloc((void**)&yk2_gpu, nn * sizeof(double));
    hipMalloc((void**)&r0_gpu, nn * sizeof(double));
    hipMalloc((void**)&uk1_gpu, nn * sizeof(double));
    hipMalloc((void**)&uk2_gpu, nn * sizeof(double));
    hipMalloc((void**)&wk_gpu, nn * sizeof(double));
    hipMalloc((void**)&vk_gpu, nn * sizeof(double));
    hipMalloc((void**)&rk_gpu, nn * sizeof(double));
    hipMalloc((void**)&dk_gpu, nn * sizeof(double));
    hipMalloc((void**)&diaginv_gpu, nn * sizeof(double));

    hipMemcpy((void*)csrrow_gpu, A.outerIndexPtr(), (nn + 1) * sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy((void*)csrcol_gpu, A.innerIndexPtr(), nnz * sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy((void*)csrval_gpu, A.valuePtr(), nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)yc_gpu, (void*)yc, nn * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*)x_gpu, (void*)x, nn * sizeof(double), hipMemcpyHostToDevice);

    Eigen::VectorXd diagonal;
    diagonal = A.diagonal();
    hipMemcpy((void*)diaginv_gpu, (void*)diagonal.data(), nn * sizeof(double), hipMemcpyHostToDevice);
    void* d_buffer = NULL;
    bool isallocatebuffer = true;
    /*----------------------------------------------------------------*/
    //_tfqmr(csrval_gpu, csrcol_gpu, csrrow_gpu, &d_buffer, diaginv_gpu, nn, nnz, &isallocatebuffer,
    //    yc_gpu, x_gpu, yk1_gpu, yk2_gpu, r0_gpu, uk1_gpu, uk2_gpu, wk_gpu, vk_gpu, rk_gpu, dk_gpu);
    int* col_data = new int[nnz];
    int* row_data = new int[nn + 1];
    for (int i = 0; i <= nn; i++) {
        row_data[i] = A.outerIndexPtr()[i];
    }
    for (int i = 0; i < nnz; i++) {
        col_data[i] = A.innerIndexPtr()[i];
    }
    
    _amg_gmres(A.valuePtr(), col_data, row_data, nn, nnz, yc, x);
    //hipMemcpy((void*)x, x_gpu, nn * sizeof(double), hipMemcpyDeviceToHost);
    /*free memory*/
    hipFree(csrval_gpu);
    hipFree(csrcol_gpu);
    hipFree(csrrow_gpu);
    hipFree(diaginv_gpu);
    hipFree(x_gpu);
    hipFree(yc_gpu);
    hipFree(yk1_gpu);
    hipFree(yk2_gpu);
    hipFree(r0_gpu);
    hipFree(uk1_gpu);
    hipFree(uk2_gpu);
    hipFree(wk_gpu);
    hipFree(vk_gpu);
    hipFree(rk_gpu);
    hipFree(dk_gpu);
    hipFree(d_buffer);
}
